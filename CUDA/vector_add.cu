#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(int* A, int* B, int* C, int N) {
    int id = threadIdx.x;
    C[id] = A[id] + B[id];
}



int main( int argc, char** argv )
{
    int N = 3;
    int a[N] = {7, 2 , 3};
    int b[N] = {6, 4, 5};
    int c[N];
    int* A, *B, *C;
    hipMalloc((void**)&A, sizeof(int) * N);
    hipMalloc((void**)&B, sizeof(int) * N);
    hipMalloc((void**)&C, sizeof(int) * N);

    hipMemcpy(A, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(B, b, sizeof(int) * N, hipMemcpyHostToDevice);

    vector_add<<<1, 3>>>(A, B, C, N);

    hipMemcpy(c, C, sizeof(int) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; ++i) {
        printf("Index %d is %d\n", i, c[i]);
    }

    hipFree(A);hipFree(B);hipFree(C);

    hipDeviceSynchronize();

    return 0;
}
