
#include <hip/hip_runtime.h>
__global__ void dot_product(int* A, int* B, int* C, int N) {
    __shared__ int tmp[3];
    int i = threadIdx.x;
    tmp[i] = A[i] * B[i];

    __syncthreads();
    int sum = 0;
    if (i == 0) {
        for (int j = 0; j < 3; ++j) sum += tmp[j];

        *C = sum;
    }
    // split the data into smaller tiles.
    // 
}
